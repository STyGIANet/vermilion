#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>


#define THREADS_PER_BLOCK 1024

__global__ void graphMatchingGPU(int* matrix, int* inOutMatching, int* outInMatching, int numNodes, int numMatchings) {
    int matchIdx = blockIdx.x * blockDim.x + threadIdx.x; // Thread index
    if (matchIdx >= numMatchings) return;

    for (int j = 0; j < numNodes; j++) {
        bool matched = false;
        for (int l = 0; l < numNodes && !matched; l++) {
            if (matrix[j * numNodes + l] == 0) continue; // Skip if there is no edge
            // Use atomicCAS to ensure thread safety for matching
            if (atomicCAS(&inOutMatching[matchIdx * numNodes + l], -1, j) == -1) {
                outInMatching[matchIdx * numNodes + j] = l;
                matched = true;
                atomicSub(&matrix[j * numNodes + l], 1);
            }
        }
    }
}

bool readMatrixFromFile(const std::string& filename, int* matrix, int numNodes) {
    std::ifstream infile(filename);
    if (!infile.is_open()) {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        return false;
    }

    std::string line;
    int row = 0;

    while (std::getline(infile, line)) {
        std::istringstream iss(line);
        int col = 0;
        int value;

        while (iss >> value) {
            if (col >= numNodes || row >= numNodes) {
                std::cerr << "Error: Matrix in file exceeds expected dimensions" << std::endl;
                return false;
            }
            matrix[row * numNodes + col] = value;
            col++;
        }

        if (col != numNodes) {
            std::cerr << "Error: Matrix row " << row << " has incorrect number of columns" << std::endl;
            return false;
        }

        row++;
    }

    if (row != numNodes) {
        std::cerr << "Error: Matrix file has incorrect number of rows" << std::endl;
        return false;
    }

    return true;
}

// // Check how many threads we are allowed to use per block
// void checkMaxThreadsPerBlock() {
//     int deviceCount;
//     cudaGetDeviceCount(&deviceCount);

//     for (int device = 0; device < deviceCount; ++device) {
//         cudaDeviceProp deviceProp;
//         cudaGetDeviceProperties(&deviceProp, device);

//         std::cout << "Device " << device << ": " << deviceProp.name << "\n";
//         std::cout << "  Maximum threads per block: " << deviceProp.maxThreadsPerBlock << "\n";
//         std::cout << "  Maximum block dimensions: (" 
//                   << deviceProp.maxThreadsDim[0] << ", " 
//                   << deviceProp.maxThreadsDim[1] << ", " 
//                   << deviceProp.maxThreadsDim[2] << ")\n";
//         std::cout << "  Maximum grid dimensions: (" 
//                   << deviceProp.maxGridSize[0] << ", " 
//                   << deviceProp.maxGridSize[1] << ", " 
//                   << deviceProp.maxGridSize[2] << ")\n";
//         std::cout << std::endl;
//     }
// }

// Matching decomposition
void runGraphMatchingGPU(int numNodes, int k, std::string filename, bool warmup, bool printMatchings) {
    int numMatchings = k * numNodes;
    size_t matrixSize = numNodes * numNodes * sizeof(int);
    size_t mappingSize = numMatchings * numNodes * sizeof(int);

    // Allocate memory on the host
    int* h_matrix = (int*)malloc(matrixSize);
    int* h_inOutMatching = (int*)malloc(mappingSize);
    int* h_outInMatching = (int*)malloc(mappingSize);

    // if (!readMatrixFromFile(filename, h_matrix, numNodes)) {
    //     std::cerr << "Matrix loading failed. Exiting.\n";
    //     free(h_matrix);
    //     return -1;
    // }

    // Initialize a permutation matrix as an example.
    for (int i = 0; i < numNodes; i++) {
        for (int j = 0; j < numNodes; j++) {
            if (j == (i+1)%numNodes){
                h_matrix[i * numNodes + j] = k*numNodes;
                // std::cout << "  Node " << i << " -> Node " << j << " traffic: " << k*numNodes << "\n";
            }
            else{
                h_matrix[i * numNodes + j] = 0;
            }
        }
    }

    std::fill_n(h_inOutMatching, numMatchings * numNodes, -1);
    std::fill_n(h_outInMatching, numMatchings * numNodes, -1);

    // Allocate memory on the device
    int* d_matrix;
    int* d_inOutMatching;
    int* d_outInMatching;
    hipMalloc(&d_matrix, matrixSize);
    hipMalloc(&d_inOutMatching, mappingSize);
    hipMalloc(&d_outInMatching, mappingSize);

    // Copy data from host to device
    hipMemcpy(d_matrix, h_matrix, matrixSize, hipMemcpyHostToDevice);
    hipMemcpy(d_inOutMatching, h_inOutMatching, mappingSize, hipMemcpyHostToDevice);
    hipMemcpy(d_outInMatching, h_outInMatching, mappingSize, hipMemcpyHostToDevice);

    // Timing using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch kernel
    int blocks = (numMatchings + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipEventRecord(start);
    graphMatchingGPU<<<blocks, THREADS_PER_BLOCK>>>(d_matrix, d_inOutMatching, d_outInMatching, numNodes, numMatchings);
    hipEventRecord(stop);

    // Synchronize and measure elapsed time
    hipEventSynchronize(stop);
    float elapsedTimeMs;
    hipEventElapsedTime(&elapsedTimeMs, start, stop);

    // Copy results back to host
    hipMemcpy(h_inOutMatching, d_inOutMatching, mappingSize, hipMemcpyDeviceToHost);
    hipMemcpy(h_outInMatching, d_outInMatching, mappingSize, hipMemcpyDeviceToHost);

    // Convert milliseconds to microseconds for better precision
    long long elapsedTimeNs = static_cast<long long>(elapsedTimeMs * 1e6);

    if (!warmup){
        std::cout << "NumToRs " << numNodes <<  " TimeNS: " << elapsedTimeNs << std::endl;
        if (printMatchings){
            std::cout << "Matchings:\n";
            for (int i = 0; i < numMatchings; i++) {
                std::cout << "Matching " << i + 1 << ":\n";
                for (int j = 0; j < numNodes; j++) {
                    int target = h_outInMatching[i * numNodes + j];
                    std::cout << "  Node " << j << " -> Node " << target << "\n";
                }
                std::cout << std::endl;
            }
        }
    }


    // Free device memory
    hipFree(d_matrix);
    hipFree(d_inOutMatching);
    hipFree(d_outInMatching);

    // Free host memory
    free(h_matrix);
    free(h_inOutMatching);
    free(h_outInMatching);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main(int argc, char* argv[]) {
    // if (argc < 3) {
    //     std::cerr << "Usage: " << argv[0] << " <numNodes> <k> [filename]\n";
    //     return -1;
    // }

    // int numNodes = std::stoi(argv[1]);
    // int k = std::stoi(argv[2]);
    // std::string filename = (argc > 3) ? argv[3] : "perm-matrix.txt";

    int k = 3;
    std::string filename = "perm-matrix.txt";

    int numExponents = 10;
    int numNodes = 2;
    
    // checkMaxThreadsPerBlock();
    
    // Warumup
    for (int i = 0; i< 100; i++)
        runGraphMatchingGPU(numNodes, k, filename, true, false);

    for (int i = 1; i <= numExponents; i++) {
        runGraphMatchingGPU(numNodes, k, filename, false, false);
        numNodes = numNodes * 2;
    }
    return 0;
}
